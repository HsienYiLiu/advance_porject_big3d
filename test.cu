
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#define EXIT_FAILURE 1
#define X 0
#define Y 1
#define Z 2
#define MAX_INT   2147483647 
//typedef enum { FALSE, TRUE } bool;

#define DIM 3                  /* Dimension of points */
typedef int    tPointi[DIM];   /* Type integer point */
typedef double tPointd[DIM];   /* Type double point */
#define PMAX 1000000             /* Max # of pts */
typedef enum boolean{ FALSE, TRUE } boolean;
tPointd Vertices[PMAX];        /* All the points */
tPointi Faces[PMAX];           /* Each triangle face is 3 indices */
tPointd com_Vertices[PMAX];
tPointi com_Faces[PMAX];
int check = 0;
tPointi Box[PMAX][2];          /* Box around each face */
int n_facets, n_vertices;      /* Original polyhedron*/
int com_facets, com_vertices,counter;  /* Original polyhedron*/
int final[PMAX];

void read_ori(void);
void read_com(void);
int ComputeBox( int F, tPointd bmin, tPointd bmax );
int irint( double x );
__device__ char BoxTest ( int n, tPointd a, tPointd b, tPointi Box );
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax );
void RandomRay( tPointd ray, int radius );
void AddVec( tPointd q, tPointd ray );
int InPolyhedron(int index, int F,int n, tPointd q, tPointd bmin, tPointd bmax, int radius );
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
int main(){
    int n, F, i;
    tPointd q, bmin, bmax;
    int radius;
    read_ori();
    read_com();
    n = n_vertices;
    F = n_facets;
    // Allocate memory
    for ( i = 0; i < DIM; i++ ){
        bmin[i] = bmax[i] = Vertices[0][i];
    }
    radius = ComputeBox( n, bmin, bmax );
    counter = com_vertices - 1;
    while( counter >= 0 ) {
        int index = com_vertices - counter - 1;
        q[X] = com_Vertices[counter][X];
        q[Y] = com_Vertices[counter][Y];
        q[Z] = com_Vertices[counter][Z];
        printf( "\n %d -------->q = %lf %lf %lf\n", counter, q[X], q[Y], q[Z] );
        printf( "In = %d\n", InPolyhedron( index , F,n, q, bmin, bmax, radius ) );
        counter--;
    }
    // check segment
    
    
    return 0;
}
__device__ double Dot( tPointd a, tPointd b )
{
    int i;
    double sum = 0.0;
    for( i = 0; i < DIM; i++ )
       sum += a[i] * b[i];

    return  sum;
}
__device__ int PlaneCoeff(tPointd N)
{
    int i;
    double t;              /* Temp storage */
    double biggest = 0.0;  /* Largest component of normal vector. */
    int m = 0;             /* Index of largest component. */


    /* Find the largest component of N. */
    for ( i = 0; i < DIM; i++ ) {
      t = fabs( N[i] );
      if ( t > biggest ) {
        biggest = t;
        m = i;
      }
    }
    return m;
}
__device__ int SegPlaneInt(double D,double denom, double num, tPointd q, tPointd r)
{
    double t;
    
    //printf("SegPlaneInt: num=%lf, denom=%lf\n", q[0], q[1] );

    if ( denom == 0.0 ) {  /* Segment is parallel to plane. */
       if ( num == 0.0 )   /* q is on plane. */
           return 5;
       else
           return 0;
    }
    else
       t = num / denom;
    //printf("SegPlaneInt: t=%lf \n", t );
    
    /*for( i = 0; i < DIM; i++ ){
       p[i] = q[i] + t * ( r[i] - q[i] );
    }*/

    if ( (0.0 < t) && (t < 1.0) )
         //return '1';
         return 9;
    else if ( num == 0.0 )   //t == 0 
         return 6;
    else if ( num == denom ) //t == 1 
         return 7;
    else return 8;
}
__device__ int AreaSign( tPointd a, tPointd b, tPointd c )  
{
    double area2;

    area2 = ( b[0] - a[0] ) * ( c[1] - a[1] ) -
            ( c[0] - a[0] ) * ( b[1] - a[1] );

    /* The area should be an integer. */
    if      ( area2 >  0.5 ) return  1;
    else if ( area2 < -0.5 ) return -1;
    else                     return  0;
} 
__device__ int InTri2D( int area0, int area1, int area2 )
{
   /* compute three AreaSign() values for pp w.r.t. each edge of the face in 2D */

   if ( ( area0 == 0 ) && ( area1 > 0 ) && ( area2 > 0 ) ||
        ( area1 == 0 ) && ( area0 > 0 ) && ( area2 > 0 ) ||
        ( area2 == 0 ) && ( area0 > 0 ) && ( area1 > 0 ) ) 
     return 2;

   if ( ( area0 == 0 ) && ( area1 < 0 ) && ( area2 < 0 ) ||
        ( area1 == 0 ) && ( area0 < 0 ) && ( area2 < 0 ) ||
        ( area2 == 0 ) && ( area0 < 0 ) && ( area1 < 0 ) )
     return 2;                 
   
   if ( ( area0 >  0 ) && ( area1 > 0 ) && ( area2 > 0 ) ||
        ( area0 <  0 ) && ( area1 < 0 ) && ( area2 < 0 ) )
     return 3;

   if ( ( area0 == 0 ) && ( area1 == 0 ) && ( area2 == 0 ) )
     //printf( "Error in InTriD\n" ); exit(EXIT_FAILURE);
     return -3;    

   if ( ( area0 == 0 ) && ( area1 == 0 ) ||
        ( area0 == 0 ) && ( area2 == 0 ) ||
        ( area1 == 0 ) && ( area2 == 0 ) )
     return 1;

   else  
     return 4;  
}
__device__ int VolumeSign( tPointd a, tPointd b, tPointd c, tPointd d )
{ 
   double vol;
   double ax, ay, az, bx, by, bz, cx, cy, cz, dx, dy, dz;
   double bxdx, bydy, bzdz, cxdx, cydy, czdz;

   ax = a[X];
   ay = a[Y];
   az = a[Z];
   bx = b[X];
   by = b[Y];
   bz = b[Z];
   cx = c[X]; 
   cy = c[Y];
   cz = c[Z];
   dx = d[X];
   dy = d[Y];
   dz = d[Z];
   //printf("%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf \n",ax,ay,az,bx,by,bz,cx,cy,cz,dx);

   bxdx=bx-dx;
   bydy=by-dy;
   bzdz=bz-dz;
   cxdx=cx-dx;
   cydy=cy-dy;
   czdz=cz-dz;
   vol =   (az-dz) * (bxdx*cydy - bydy*cxdx)
         + (ay-dy) * (bzdz*cxdx - bxdx*czdz)
         + (ax-dx) * (bydy*czdz - bzdz*cydy);


   /* The volume should be an integer. */
   if      ( vol > 0.5 )   return  1;
   else if ( vol < -0.5 )  return -1;
   else                    return  0;
}
__device__ int SegTriCross(int vol0, int vol1, int vol2)
{
   
 
   //printf( "SegTriCross:  vol0 = %d; vol1 = %d; vol2 = %d\n", vol0, vol1, vol2 ); 
     
   /* Same sign: segment intersects interior of triangle. */
   if ( ( ( vol0 > 0 ) && ( vol1 > 0 ) && ( vol2 > 0 ) ) || 
        ( ( vol0 < 0 ) && ( vol1 < 0 ) && ( vol2 < 0 ) ) )
      return 33;
   
   /* Opposite sign: no intersection between segment and triangle */
   if ( ( ( vol0 > 0 ) || ( vol1 > 0 ) || ( vol2 > 0 ) ) &&
        ( ( vol0 < 0 ) || ( vol1 < 0 ) || ( vol2 < 0 ) ) )
      return 0;

   else if ( ( vol0 == 0 ) && ( vol1 == 0 ) && ( vol2 == 0 ) )
     //fprintf( stderr, "Error 1 in SegTriCross\n" ), exit(EXIT_FAILURE);
     return -3;   
 
   /* Two zeros: segment intersects vertex. */
   else if ( ( ( vol0 == 0 ) && ( vol1 == 0 ) ) || 
             ( ( vol0 == 0 ) && ( vol2 == 0 ) ) || 
             ( ( vol1 == 0 ) && ( vol2 == 0 ) ) )
      return 11;

   /* One zero: segment intersects edge. */
   else if ( ( vol0 == 0 ) || ( vol1 == 0 ) || ( vol2 == 0 ) )
      return 22;
   
   else
     return -3;
     //fprintf( stderr, "Error 2 in SegTriCross\n" ), exit(EXIT_FAILURE);
}
__global__ void check_each( tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V,int F,tPointi * ori_F,tPointd * ori_V,tPointd * r,tPointd * q, tPointi *Box, int * out)
{
      
      volatile __shared__ bool FoundIt;
      // initialize shared status
      FoundIt = false;
      __syncthreads();
      int k = 0, crossings = 0;
      int code = -1; 
      //int i = blockIdx.x;
      int i = blockIdx.x;
      //printf("i %d \n",i);
      crossings = 0;
      // get N
      tPointd N,rq;
      //printf("q qqqqqq %lf, %lf, %lf\n",q[0][X],q[0][Y],q[0][Z]);
      N[X] = (ori_V[ori_F[i][Z]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Y]][Y]-ori_V[ori_F[i][X]][Y])-(ori_V[ori_F[i][Y]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y]);
      N[Y] = (ori_V[ori_F[i][Y]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Z])-(ori_V[ori_F[i][Y]][X]- ori_V[ori_F[i][X]][X])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y]);
      N[Z] = (ori_V[ori_F[i][Y]][X]- ori_V[ori_F[i][X]][X])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y])-(ori_V[ori_F[i][Y]][Y]- ori_V[ori_F[i][X]][Y])*(ori_V[ori_F[i][Z]][X]- ori_V[ori_F[i][X]][X]);
      // Cal dot
      double D,num,denom;
      D = Dot( ori_V[ori_F[i][0]], N );
      int m;
      m = PlaneCoeff(N);
      num = D - Dot( *q, N );
      rq[X] = r[0][X] - q[0][X];
      rq[Y] = r[0][Y] - q[0][Y];
      rq[Z] = r[0][Z] - q[0][Z];
      denom = Dot(rq,N);
      int tmp_code = SegPlaneInt(D, denom, num, *q, *r);
      //double t = num / denom;

      //printf("SegPlaneInt: %d\n", tmp_code );      
      //printf("bmax: %lf,%lf,%lf\n", bmax[0][0],bmax[0][1],bmax[0][2] );      
      //f = &Box[0][0][0];
      //tmp_code = 1;
      if(i < F){
         if ( !InBox( *q, *bmin, *bmax ) == 2 ){
              out[i] = 0;
              FoundIt = true;
              //printf("wpwowow %d\n", out[i]);
         }
         /*if (BoxTest( i, *q, *r, *Box ) == '0' && FoundIt == false) {
              
              out[i] = 0;
              FoundIt = true;
              printf("BoxTest = 0!\n");
         }else */
         if(FoundIt == false){
             if(tmp_code == 8){
                 tmp_code = 8;
                 //FoundIt == true;
             }
             if(tmp_code == 6){
                 tPointd pp,Tp[3];     // projected T: three new vertices 
                 //t = num / denom;
                 
                 // Project out coordinate m in both p and the triangular face 
                 int j = 0;
                 for ( i = 0; i < DIM; i++ ) {
                     if ( i != m ) {    //skip largest coordinate 
                         pp[j] = q[0][i];
                         for ( k = 0; k < 3; k++ ){
	                     Tp[k][j] = ori_V[ori_F[i][k]][i];
                             //printf(" plane=(%lf)\n", Tp[k][j]);
                         }
                         j++;
                          
                      }
                 }
                 int area0 = AreaSign( pp, Tp[0], Tp[1] );
                 int area1 = AreaSign( pp, Tp[1], Tp[2] );
                 int area2 = AreaSign( pp, Tp[2], Tp[0] );                 
                 tmp_code = InTri2D(  area0, area1, area2 );
                 //FoundIt == true;
                 //printf("areaaa %d\n", out[i]);
                 //code = InTri2D( Tp, pp );
             }
             else if(tmp_code == 7){
                 tPointd pp,Tp[3];     // projected T: three new vertices 
                 //t = num / denom;

                 // Project out coordinate m in both p and the triangular face 
                 int j = 0;
                 for ( i = 0; i < DIM; i++ ) {
                     if ( i != m ) {    //skip largest coordinate 
                         pp[j] = r[0][i];
                         for ( k = 0; k < 3; k++ ){
                             Tp[k][j] = ori_V[ori_F[i][k]][i];
                             //printf(" plane=(%lf)\n", Tp[k][j]);
                         }
                         j++;

                      }
                 }
                 int area0 = AreaSign( pp, Tp[0], Tp[1] );
                 int area1 = AreaSign( pp, Tp[1], Tp[2] );
                 int area2 = AreaSign( pp, Tp[2], Tp[0] );
                 tmp_code = InTri2D(  area0, area1, area2 );
                 //FoundIt == true;
                 //printf("areaaa %d\n", out[i]);
                 //code = InTri2D( Tp, pp );
             //}else if(tmp_code == 10){
                 //out[i] = 10;
                 //FoundIt == true;
             }else if(tmp_code == 9){
                 int vol0, vol1, vol2;
                 vol0 = VolumeSign( q[0], ori_V[ori_F[i][0] ], ori_V[ori_F[i][1] ], r[0] );
                 vol1 = VolumeSign( q[0], ori_V[ori_F[i][1] ], ori_V[ori_F[i][2] ], r[0] );
                 vol2 = VolumeSign( q[0], ori_V[ori_F[i][2] ], ori_V[ori_F[i][0] ], r[0] );
                 //printf( "SegTriCross:  vol0 = %d; vol1 = %d; vol2 = %d\n", vol0, vol1, vol2 ); 
                 tmp_code = SegTriCross(vol0,vol1,vol2);
                 //FoundIt = true;

             }else{
                 tmp_code = tmp_code;
         
             }
         }
         if(FoundIt == false){
           code = tmp_code;
           //code = 10;
           printf( "Face = %d: BoxTest/SegTriInt returns %d\n\n", i, code ); 
          
           //If ray is degenerate, then goto outer while to generate another.
           //if ( code == 10 || code == 2 || code == 3 ) {
           if( code == 5 || code == 11 || code == 22){
              printf("Degenerate ray\n");
              //out[i] = -3;
              //crossing++;
              FoundIt = true;  
              //printf("out %d\n",out[i]);
           }
         
           //If ray hits face at interior point, increment crossings.
           else if ( code == 33) {
              crossings++;
              //printf( "crossings = %d\n", crossings );
           }

           //If query endpoint q sits on a V/E/F, return that code.
           else if ( code == 2 || code == 3|| code == 1)
              //return code;
              out[i] = code;

           //If ray misses triangle, do nothing. 
           else if ( code == 44 || code == 8|| code == 8 )
              ;

           else{
              out[i] = -3;
           } 
              //fprintf( stderr, "Error, exit(EXIT_FAILURE)\n" ), exit(1);      
         }
         //if( ( crossings % 2 ) == 1 )
            //out[i] = 1;
         out[i] = crossings;
         printf("check if every point is check i -> %d, out -> %d \n",i,out[i]);
         }
}

int InPolyhedron(int index, int F,int n, tPointd q, tPointd bmin, tPointd bmax, int radius )
{
    tPointd r;  /* Intersection point; not used. */
    int k = 0;
    tPointd *d_bmin, *d_bmax, *c_com_V,*ori_V,*final_r,*final_q;
    tPointi *cu_box,*ori_F;
    int *out,*result;
    //char result[counter];
    result = (int *)malloc(sizeof(int)*F);
    gpuErrchk(hipMalloc(&c_com_V,sizeof(tPointd)*F));
    hipMalloc(&ori_V,sizeof(tPointd)*n);
    hipMalloc(&ori_F,sizeof(tPointi)*F);
    hipMalloc(&d_bmax,sizeof(tPointd)*3);
    hipMalloc(&d_bmin,sizeof(tPointd)*3);
    hipMalloc(&final_r,sizeof(tPointd)*3);
    hipMalloc(&final_q,sizeof(tPointd)*3); 
    hipMalloc(&cu_box,sizeof(tPointi)*2*F);
    hipMalloc(&out,sizeof(tPointi)*F);

    hipMemcpy(c_com_V, com_Vertices, sizeof(tPointd)*F, hipMemcpyHostToDevice);
    hipMemcpy(ori_V, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice);
    hipMemcpy(ori_F, Faces, sizeof(tPointi)*F, hipMemcpyHostToDevice);
    hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(final_q, q, sizeof(tPointd)*3, hipMemcpyHostToDevice);
    hipMemcpy(cu_box, Box, sizeof(tPointi)*2*F, hipMemcpyHostToDevice);
    hipMemcpy(out, result, sizeof(int)*F, hipMemcpyHostToDevice);

    //printf("Box test %d\n",cu_box[0][0][0]);
   
   //LOOP:
    while( k++ < F) {
      //crossings = 0;
  
      RandomRay( r, radius ); 
      AddVec( q, r ); // add the ray with the point to create end point
      
      printf("Ray endpoint: (%lf,%lf,%lf)\n", r[0],r[1],r[2] );
      hipMemcpy(final_r, r, sizeof(tPointd)*3, hipMemcpyHostToDevice);
      check_each<<<F, 1>>>(d_bmin,d_bmax,radius,c_com_V,F,ori_F, ori_V,final_r,final_q,cu_box, out);     
      hipMemcpy(result,out, sizeof(int)*F, hipMemcpyDeviceToHost);
      //printf("RRResult %d\n",k);   
      break;

   } 
   // check result
   int final_result = 0; 
   for(int c = 0; c < counter; c++){
       final_result = final_result + result[c];
   }
   if(final_result % 2 == 1){
       final_result = 1;
   }else{
       final_result = 0;
   }
   //printf("testt final %d\n", index);
   free(result);
   hipFree(d_bmin);hipFree(d_bmax);hipFree(c_com_V);
   hipFree(ori_F);hipFree(ori_V);hipFree(final_r);
   hipFree(final_q);hipFree(out);hipFree(cu_box);
   return final_result;
}
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax )
{
  //printf("baxxxx test %lf, %lf, %lf\n", q[Z],bmax[Z],q[X]);
  if( ( bmin[X] <= q[X] ) && ( q[X] <= bmax[X] ) &&
      ( bmin[Y] <= q[Y] ) && ( q[Y] <= bmax[Y] ) &&
      ( bmin[Z] <= q[Z] ) && ( q[Z] <= bmax[Z] ) ){
    //printf("TRUEEEE");
    return 1;
   
  }
  return 2;
}
/* Return a random ray endpoint */
 void RandomRay( tPointd ray, int radius )
{
  double x, y, z, w, t;
  /* Generate a random point on a sphere of radius 1. */
  /* the sphere is sliced at z, and a random point at angle t
     generated on the circle of intersection. */
  z = 2.0 * (double) rand() / MAX_INT - 1.0;
  t = 2.0 * M_PI * (double) rand() / MAX_INT;
  //printf("check %lf\n",rand1);
  w = sqrt( 1 - z*z );
  x = w * cos( t );
  y = w * sin( t );
  
  ray[X] = radius * x;
  ray[Y] = radius * y;
  ray[Z] = radius * z;
  
  /*printf( "RandomRay returns %6d %6d %6d\n", ray[X], ray[Y], ray[Z] );*/
}
void AddVec( tPointd q, tPointd ray )
{
  int i;
  
  for( i = 0; i < DIM; i++ )
    ray[i] = q[i] + ray[i];
}
__device__ char BoxTest ( int n, tPointd a, tPointd b, tPointi Box)
{
   int i; /* Coordinate index */
   int w;
   //printf(" Box %d\n", Box[0][0][0]);
   for ( i=0; i < DIM; i++ ) {
       w = Box[n]; //min: lower left 
       if ( ((int)a[i] < w ) && ((int)b[i] < w) ) return '0';
       w = Box[n]; // max: upper right 
       if ( ((int)a[i] > w) && ((int)b[i] > w) ) return '0';
   }
   return '?';
}
__global__ void cal(tPointd *bmin, tPointd *bmax,tPointd *V,int F){

    int i = blockIdx.x; // will give you X block Index at that particular thread
    int j = blockIdx.y; // will give you Y block Index at that particular thread. 
    if(i < F){
        //j = j%3;
        for(j = 0; j < 3; j++){
            if( V[i][j] < *bmin[j] )
                *bmin[j] = V[i][j];
            if( V[i][j] > *bmax[j] ){
                *bmax[j] = V[i][j];
                //printf("V %lf\n",V[i][j]);
            }
 //           printf("Check i = %d, j = %d, F = %d\n",i,j,F);
        }
    }
   // printf("bmax %lf, bmin %lf \n",*bmax[Y],*bmin[Y]);
}
int ComputeBox( int n, tPointd bmin, tPointd bmax ){
  double radius;
  tPointd *min, *max,*d_bmax, *d_bmin, *d_a;
  min = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array2 on host 
  max = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array2 on host 

  hipMalloc(&d_a,sizeof(tPointd)*n);
  hipMalloc(&d_bmax,sizeof(tPointd)*3);
  hipMalloc(&d_bmin,sizeof(tPointd)*3);

  hipMemcpy(d_a, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
  hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);

  //dim3 blockSize(256);
  //dim3 gridSize((n + blockSize.x) / blockSize.x);
  cal<<<n+1, 1>>>(d_bmin, d_bmax, d_a, n);
  hipMemcpy(max,d_bmax, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  hipMemcpy(min,d_bmin, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  printf("------------------------\n");
  printf("bmax %lf bmin %lf \n",*max[X],*min[X]);
  printf("bmax %lf, bmin %lf \n",*max[Y],*min[Y]);
  printf("bmax %lf, bmin %lf \n",*max[Z],*min[Z]);
  radius = sqrt( pow( (double)(*max[X] - *min[X]), 2.0 ) +
                 pow( (double)(*max[Y] - *min[Y]), 2.0 ) +
                 pow( (double)(*max[Z] - *min[Z]), 2.0 ) );
  printf("radius = %lf\n", radius);
  bmax[0] = *max[0];bmax[1] = *max[1];bmax[2] = *max[2];
  bmin[0] = *min[0];bmin[1] = *min[1];bmin[2] = *min[2];
  
  //printf("radius = %lf\n", bmax[2]);
  hipFree(d_bmax);
  hipFree(d_bmin);
  hipFree(d_a);
  free(max);
  free(min);

  return irint( radius +1 ) + 1;
}

int irint( double x )
{
        return (int) rint( x );
}
void read_ori(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    //float a,b,c;
    fp = fopen("b.off", "r");
    int i = 0;
    int j,k,w;

    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL ) {
            // init facets and vertices
            if(count <= 2){
                if(token_count == 0){
                    n_vertices = atoi(token);
                }else if(token_count == 1){
                    n_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
                if(token_count == 0){
                    Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    Vertices[count - 4][Y] = atof(token);
                }else{
                    Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            } else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;

                if(token_count == 1){
                    Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    Faces[i][Y] = atoi(token);
                    //printf("->>>>  %d\n",Faces[count - 144][X]);
                }else if(token_count == 3){
                    Faces[i][Z] = atoi(token);
                    for ( j=0; j < 3; j++ ) {
                        Box[i][0][j] = Vertices[ Faces[i][0] ][j];
                        Box[i][1][j] = Vertices[ Faces[i][0] ][j];
                  }

               for ( k=1; k < 3; k++ )
               for ( j=0; j < 3; j++ ) {
                  w = Vertices[ Faces[i][k] ][j];
                  //printf("->>>>  %d\n",Faces[i][k]);
                  if ( w < Box[i][0][j] ) Box[i][0][j] = w;
                  if ( w > Box[i][1][j] ) Box[i][1][j] = w;
               }
               /*
               printf("Bounding box: (%d,%d,%d);(%d,%d,%d)\n",
                  Box[i][0][0],
                  Box[i][0][1],
                  Box[i][0][2],
                  Box[i][1][0],
                  Box[i][1][1],
                  Box[i][1][2] );
                */
                }
                token_count++;
            }
            token = strtok(NULL," ");
            //free(token);
        }
    }
    if (line)
        free(line);
}
void read_com(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    //float a,b,c;
    fp = fopen("small.off", "r");
    int i ;
    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL) {
            // init facets and vertices
            if(count <= 2){
                printf("setting of file  %s\n", token);
                if(token_count == 0){
                    com_vertices = atoi(token);
                }else if(token_count == 1){
                    com_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
               if(token_count == 0){
                    com_Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    com_Vertices[count - 4][Y] = atof(token);
                }else{
                    com_Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            }else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;
                if(token_count == 1){
                    com_Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    com_Faces[i][Y] = atoi(token);
                }else if(token_count == 3){
                    com_Faces[i][Z] = atoi(token);
                }
                token_count++;
            }
            token = strtok(NULL, " ");
        }
    }
    if (line)
        free(line);
}
